#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdint.h>
#include <iostream>

#include "ndi_convert_interface.h"

#ifdef __cplusplus
#define EXTERNC extern "C" __declspec(dllexport)
#else
#define EXTERNC
#endif

#define THREADS 256

__device__ uint8_t clampUint8(int v)
{
    if (v > 255)
        return 255;
    if (v < 0)
        return 0;
    return (uint8_t)v;
}

__global__ void kernelUYVYToRGBA(uint8_t *d_src, uint8_t *d_dest, int pixcount)
{
    int pix = blockIdx.x * blockDim.x + threadIdx.x;

    if (pix >= pixcount)
        return;
    int i = pix * 2;

    int y, u, v;
    y = d_src[i + 1] - 16;

    if (pix % 2 == 0)
    {
        u = d_src[i];
        v = d_src[i + 2];
    }
    else
    {
        v = d_src[i];
        u = d_src[i - 2];
    }
    u -= 128;
    v -= 128;

    uint8_t r = clampUint8((int)roundf(1.164 * y + 1.596 * v));
    uint8_t g = clampUint8((int)roundf(1.164 * y - 0.392 * u - 0.813 * v));
    uint8_t b = clampUint8((int)roundf(1.164 * y + 2.017 * u));

    int offset = pix * 4;
    d_dest[offset] = r;
    d_dest[offset + 1] = g;
    d_dest[offset + 2] = b;
    d_dest[offset + 3] = 255;
}

EXTERNC void UYVYToRGBA(int width, int height, uint8_t *src, uint8_t *dest)
{
    uint8_t *d_src;
    uint8_t *d_dest;
    size_t srcSize = sizeof(uint8_t) * width * height * 2;
    size_t destSize = sizeof(uint8_t) * width * height * 4;
    int pixcount = width * height;

    hipMalloc(&d_src, srcSize);
    hipMemcpy(d_src, src, srcSize, hipMemcpyHostToDevice);

    hipMalloc(&d_dest, destSize);
    int blockCount = (int)ceil(pixcount / (double)THREADS);

    kernelUYVYToRGBA<<<blockCount, THREADS>>>(d_src, d_dest, pixcount);
    hipDeviceSynchronize();

    hipMemcpy(dest, d_dest, destSize, hipMemcpyDeviceToHost);

    hipFree(d_src);
    hipFree(d_dest);
}

__global__ void kernelBGRAToRGBA(uint8_t *d_src, uint8_t *d_dest, int pixcount)
{
    int pix = blockIdx.x * blockDim.x + threadIdx.x;

    if (pix >= pixcount)
        return;
    int pixb = pix * 4;

    d_dest[pixb] = d_src[pixb + 2];
    d_dest[pixb + 1] = d_src[pixb + 1];
    d_dest[pixb + 2] = d_src[pixb];
    d_dest[pixb + 3] = d_src[pixb + 3];
    // d_dest[pixb] = (uint8_t)255;
    // d_dest[pixb + 1] = (uint8_t)255;
    // d_dest[pixb + 2] = (uint8_t)0;
    // d_dest[pixb + 3] = (uint8_t)255;
}

EXTERNC void BGRAToRGBA(int width, int height, uint8_t *src, uint8_t *dest)
{
    uint8_t *d_src;
    uint8_t *d_dest;
    size_t srcSize = sizeof(uint8_t) * width * height * 4;
    size_t destSize = sizeof(uint8_t) * width * height * 4;
    int pixcount = width * height;

    hipMalloc(&d_src, srcSize);
    hipMemcpy(d_src, src, srcSize, hipMemcpyHostToDevice);

    hipMalloc(&d_dest, destSize);
    int blockCount = (int)ceil(pixcount / (double)THREADS);

    kernelBGRAToRGBA<<<blockCount, THREADS>>>(d_src, d_dest, pixcount);
    hipDeviceSynchronize();

    hipMemcpy(dest, d_dest, destSize, hipMemcpyDeviceToHost);

    hipFree(d_src);
    hipFree(d_dest);
}

EXTERNC void getDeviceProperties(int *major, int *minor)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    major[0] = deviceProp.major;
    minor[0] = deviceProp.minor;
}
